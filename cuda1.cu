#include "hip/hip_runtime.h"
// #include<stdio.h>
// #include<hip/hip_runtime.h>

// __global__ void arrAdd(int *x,int *y,int *z)
// {
//     int id = blockIdx.x;
//     z[id] = x[id] + y[id];
// }

// int main()
// {
//     int a[6],b[6],c[6];
//     int *d,*e,*f,i;
//     for(i=0;i<6;i++)
//     {
//         scanf("%d",&a[i]);
//     }
//     for(i=0;i<6;i++)
//     {
//         scanf("%d",&b[i]);
//     }
//     hipMalloc((void **)&d,6*sizeof(int));
//     hipMalloc((void **)&e,6*sizeof(int));
//     hipMalloc((void **)&f,6*sizeof(int));
//     hipMemcpy(d,a,6*sizeof(int),hipMemcpyHostToDevice);
//     hipMemcpy(e,b,6*sizeof(int),hipMemcpyHostToDevice);
//     arrAdd<<<6,1>>(d,e,f);
//     hipMemcpy(c,f,6*sizeof(int),hipMemcpyDeviceToHost);
//     for(i=0;i<6;i++)
//     {
//         printf("%d",c[i]);
//     }
//     hipFree(d);
//     hipFree(e);
//     hipFree(f);
//     return 0;
// }


#include<stdio.h>
#include<hip/hip_runtime.h>
#include<ctime>

__global__ void arrAdd(int *x,int *y,int *z)
{
    int id = blockIdx.x;
    z[id] = x[id] + y[id];
}

int main()
{
    int a[6],b[6],c[6];
    int *d,*e,*f;
    int size = 6*sizeof(int);
    hipMalloc((void **)&d,size);
    hipMalloc((void **)&e,size);
    hipMalloc((void **)&f,size);
    hipMemcpy(d,a,size,hipMemcpyHostToDevice);
    hipMemcpy(e,b,size,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    double elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    arrAdd<<<6,1>>>(d,e,f);
    hipEventRecord(stop);
    cudaEventSyncronize(stop);
    hipEventElapsedTime(&elapsed_time,start,stop);
    hipMemcpy(c,e,size,hipMemcpyDeviceToHost);
    printf("%d ",c[i]);
    hipFree(d);
    hipFree(e);
    hipFree(f);
    retrun 0;
}















