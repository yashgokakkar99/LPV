#include "hip/hip_runtime.h"
// #include<stdio.h>
// #include<hip/hip_runtime.h>
// #define N 3

// __global__ void matMul(int *a,int *b,int *c)
// {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     int sum = 0;
//     for(int i=0;i<N;++i)
//     {
//         sum+=a[row*N+i]*b[i*N+col];
//     }
//     c[row*N+col] = sum;
// }

// int main()
// {
//     int a[N][N],b[N][N],c[N][N];
//     int *d,*e,*f;
//     int size = N*N*sizeof(int);
//     for(int i=0;i<N;++i)
//     {
//         for(int j=0;j<N;++j)
//         {
//             scanf("%d",&a[i][j])
//         }
//     }
//     for(int i=0;i<N;++i)
//     {
//         for(int j=0;j<N;++j)
//         {
//             scanf("%d",&b[i][j])
//         }
//     }
//     hipMalloc((void **)&d,size);
//     hipMalloc((void **)&e,size);
//     hipMalloc((void **)&f,size);
//     hipMemcpy(d,a,size,hipMemcpyHostToDevice);
//     hipMemcpy(e,b,size,hipMemcpyHostToDevice);
//     dim3 threadsPerBlock(N,N);
//     dim3 numBlocks(1,1);
//     matMul<<<numBlocks,threadsPerBlock>>>(d,e,f);
//     hipMemcpy(c,f,size,hipMemcpyDeviceToHost);
//     for(int i=0;i<N;++i)
//     {
//         for(int j=0;j<N;++j)
//         {
//             printf("%d",c[i][j]);
//         }
//     }
//     return 0;
// }




#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 3

__global__ void matMul(int *a,int *b,int *c)
{
    int row = blockIdx.y* blockDim.y+ threadIdx.y;
    int col = blockIdx.x* blockDim.x+ threadIdx.x;
    int sum = 0;
    for(int i=0;i<N;i++)
    {
        sum += a[N*row+i]*b[i*N+col];
        c[N*row+col] = sum;
    }
} 

dim3 threadsPerBlock(N,N)
dim3 numBlocks(1,1)